#include "hip/hip_runtime.h"
#include <stdio.h>

#include <poisson.hpp>
#include <poisson.cuh>
#include <assertions.hpp>
#include <grid.hpp>
#include <solver.hpp>


template <typename S, typename P, typename T=double>
void convergence_test(const int num_grids, SolverOptions opts) {
        T rate = 0.0;
        T err1 = 0.0;
        T modes = 1.0;
        int l = 2;
        T h = 1.0;
        printf("MMS convergence test\n");
        {
                S tmp;
                printf("Solver: %s \n", tmp.name());
        }
        printf("Grid Size \t Iterations \t Time (ms) \t Residual \t Error \t\t Rate \n");
        for (int i = 0; i < num_grids; ++i) {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                P problem(l, h, modes);
                S solver(problem);

                hipEventRecord(start);
                SolverOutput out = solve(solver, problem, opts);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                float elapsed = 0;
                hipEventElapsedTime(&elapsed, start, stop);

                rate = log2(err1 / out.error);
                int n = (1 << l) + 1;
                printf("%4d x %-4d \t %-7d \t %-5.5f \t %-5.5g \t %-5.5g \t %-5.5f \n", 
                       n, n,
                       out.iterations, elapsed, out.residual, out.error, rate);
                err1 = out.error;
                l++;
                h /= 2;
        }
}


int main(int argc, char **argv) {

        using Number = double;
        SolverOptions opts;
        opts.verbose = 1;
        opts.info = 10;
        opts.max_iterations = 1e4;
        opts.eps = 1e-8;
        opts.mms = 1;
        int l = 4;
        int n = (1 << l) + 1;
        double h = 1.0 / (n - 1);
        double modes = 1.0;
        using Problem = Poisson<Number>;

    
        {

                using Problem = Poisson<Number>;
                Problem problem(l, h, modes);
                using Smoother=GaussSeidelRedBlack;
                Smoother solver;
                auto out = solve(solver, problem, opts);
                printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);

        }
        
        {
                Problem problem(l, h, modes);
                using Smoother=GaussSeidelRedBlack;
                using MG=Multigrid<Smoother, Problem, Number>;
                MG mg(problem);
                auto out = solve(mg, problem, opts);
                printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);

        }      

        {
                using CUDAProblem = CUDAPoisson<L1NORM, Number>;
                CUDAProblem problem(l, h, modes);
                using CUDASmoother = CUDAGaussSeidelRedBlack;

                CUDASmoother solver;
                auto out = solve(solver, problem, opts);
                printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);
        }

        {
                using CUDAProblem = CUDAPoisson<L1NORM, Number>;
                using CUDASmoother = CUDAGaussSeidelRedBlack;
                using CUDAMG = CUDAMultigrid<CUDASmoother, CUDAProblem, Number>;
                
                CUDAProblem problem(l, h, modes);
                CUDAMG solver(problem);
                auto out = solve(solver, problem, opts);
                printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);
        }

        {
                using Smoother=GaussSeidelRedBlack;
                using MG=Multigrid<Smoother, Problem, Number>;
                opts.verbose = 0;
                
                int num_refinements = 12;
                convergence_test<MG, Problem>(num_refinements, opts);
        }
        {
                using CUDAProblem = CUDAPoisson<L1NORM, Number>;
                using CUDASmoother = CUDAGaussSeidelRedBlack;
                using CUDAMG = CUDAMultigrid<CUDASmoother, CUDAProblem, Number>;
                
                opts.verbose = 0;
                int num_refinements = 12;
                convergence_test<CUDAMG, CUDAProblem>(num_refinements, opts);
        }
}
