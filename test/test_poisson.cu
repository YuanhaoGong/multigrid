#include "hip/hip_runtime.h"
#include <stdio.h>

#include <poisson.hpp>
#include <assertions.hpp>
#include <grid.hpp>
#include <solver.hpp>


template <typename S, typename P, typename T=double>
void convergence_test(const int num_grids, SolverOptions opts) {
        T rate = 0.0;
        T err1 = 0.0;
        T modes = 1.0;
        int l = 2;
        T h = 1.0;
        printf("MMS convergence test\n");
        {
                S tmp;
                printf("Solver: %s \n", tmp.name());
        }
        printf("Refinement \t Iterations \t Residual \t Error \t\t Rate \n");
        for (int i = 0; i < num_grids; ++i) {
                P problem(l, h, modes);
                S solver(problem);
                SolverOutput out = solve(solver, problem, opts);
                rate = log2(err1 / out.error);
                printf("%-7d \t %-7d \t %-5.5g \t %-5.5g \t %-5.5g \n", i,
                       out.iterations, out.residual, out.error, rate);
                err1 = out.error;
                l++;
                h /= 2;
        }
}


int main(int argc, char **argv) {

        using Number = double;
        SolverOptions opts;
        opts.verbose = 1;
        opts.info = 1;
        opts.max_iterations = 20;//e4;
        opts.eps = 1e-8;
        opts.mms = 1;
        int l = 4;
        int n = (1 << l) + 1;
        double h = 1.0 / (n - 1);
        double modes = 1.0;
        using Problem = Poisson<Number>;

        {
                Problem problem(l, h, modes);
                using Smoother=GaussSeidelRedBlack;
                using MG=Multigrid<Smoother, Problem, Number>;
                MG mg(problem);
                auto out = solve(mg, problem, opts);
                printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);

                opts.verbose = 0;
                convergence_test<MG, Problem>(10, opts);
        }

        //{
        //opts.verbose = 1;
        //Problem problem(7, h, modes);
        //GaussSeidel gs;
        //auto out = solve(gs, problem, opts);
        //printf("Iterations: %d, Residual: %g \n", out.iterations, out.residual);
        //}
        //convergence_test<GaussSeidel, Problem, Number>(problem, 3, opts);
        //convergence_test<double, GaussSeidelRedBlack>(5, opts);

}
